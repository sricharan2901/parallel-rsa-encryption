
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define MODULUS 3233 // Example modulus for RSA (should be a product of two primes)
#define PUB_EXP 17   // Example public exponent for RSA
#define PRIV_EXP 413 // Example private exponent for RSA

__device__ unsigned long long mod_exp_cuda(unsigned long long base, unsigned long long exp, unsigned long long mod) {
    unsigned long long result = 1;
    base = base % mod;
    while (exp > 0) {
        if (exp % 2 == 1) {
            result = (result * base) % mod;
        }
        exp = exp >> 1;
        base = (base * base) % mod;
    }
    return result;
}

__global__ void rsa_encrypt_kernel(unsigned char *input, unsigned long long *output, int len, unsigned long long exp, unsigned long long mod) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        output[idx] = mod_exp_cuda((unsigned long long)input[idx], exp, mod);
    }
}

__global__ void rsa_decrypt_kernel(unsigned long long *input, unsigned char *output, int len, unsigned long long exp, unsigned long long mod) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        output[idx] = (unsigned char)mod_exp_cuda(input[idx], exp, mod);
    }
}

void rsa_encrypt(unsigned char *input, unsigned long long *output, int len) {
    unsigned char *d_input;
    unsigned long long *d_output;

    hipMalloc((void **)&d_input, len * sizeof(unsigned char));
    hipMalloc((void **)&d_output, len * sizeof(unsigned long long));

    hipMemcpy(d_input, input, len * sizeof(unsigned char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (len + blockSize - 1) / blockSize;
    rsa_encrypt_kernel<<<gridSize, blockSize>>>(d_input, d_output, len, PUB_EXP, MODULUS);

    hipMemcpy(output, d_output, len * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void rsa_decrypt(unsigned long long *input, unsigned char *output, int len) {
    unsigned long long *d_input;
    unsigned char *d_output;

    hipMalloc((void **)&d_input, len * sizeof(unsigned long long));
    hipMalloc((void **)&d_output, len * sizeof(unsigned char));

    hipMemcpy(d_input, input, len * sizeof(unsigned long long), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (len + blockSize - 1) / blockSize;
    rsa_decrypt_kernel<<<gridSize, blockSize>>>(d_input, d_output, len, PRIV_EXP, MODULUS);

    hipMemcpy(output, d_output, len * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    FILE *file = fopen("input.txt", "rb");
    if (!file) {
        fprintf(stderr, "Failed to open input file\n");
        return 1;
    }

    fseek(file, 0, SEEK_END);
    int fileSize = ftell(file);
    fseek(file, 0, SEEK_SET);

    unsigned char *input = (unsigned char *)malloc(fileSize);
    unsigned long long *encrypted = (unsigned long long *)malloc(fileSize * sizeof(unsigned long long));
    unsigned char *decrypted = (unsigned char *)malloc(fileSize);

    fread(input, 1, fileSize, file);
    fclose(file);

    struct timeval start, end;

    // Encryption
    gettimeofday(&start, NULL);
    rsa_encrypt(input, encrypted, fileSize);
    gettimeofday(&end, NULL);
    printf("Encryption completed.\n");
    printf("Public Key: %llu\n", PUB_EXP);
    printf("Modulus: %llu\n", MODULUS);
    printf("Encryption Time: %.6f seconds\n", ((end.tv_sec - start.tv_sec) * 1000000 + end.tv_usec - start.tv_usec) / 1000000.0);

    // Decryption
    gettimeofday(&start, NULL);
    rsa_decrypt(encrypted, decrypted, fileSize);
    gettimeofday(&end, NULL);
    printf("Decryption completed.\n");
    printf("Private Key: %llu\n", PRIV_EXP);
    printf("Modulus: %llu\n", MODULUS);
    printf("Decryption Time: %.6f seconds\n", ((end.tv_sec - start.tv_sec) * 1000000 + end.tv_usec - start.tv_usec) / 1000000.0);

    // Save decrypted output to file
    file = fopen("decrypted_output.txt", "wb");
    fwrite(decrypted, 1, fileSize, file);
    fclose(file);

    free(input);
    free(encrypted);
    free(decrypted);

    return 0;
}
